#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define M 256  // number of rows in A and C
#define K 512  // number of columns in A and rows in B
#define N 256  // number of columns in B and C
#define BLOCK_SIZE 32

// naive matmul ofc

// m: rows in A and C, k: cols in A and rows in B, n: cols in B and C
void matmul_cpu(float *A, float *B, float *C, uint32_t m, uint32_t k, uint32_t n) {
  for (uint32_t i = 0; i < m; ++i) { // each row in C
    for (uint32_t j = 0; j < n; ++j) { // each col in C
      float sum = 0.0f;
      for (uint32_t l = 0; l < k; ++l) // each col in A, row in B
        sum += A[i * k + l] * B[l * n + j]; // mul each at the current row and col
      C[i * n + j] = sum; // set sum in output matrix
    }
  }
}

__global__ void matmul_gpu(float *A, float *B, float *C, uint32_t m, uint32_t k, uint32_t n) {
  uint32_t row = blockIdx.y * blockDim.y + threadIdx.y;
  uint32_t col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < m && col < n) {
    float sum = 0.0f;
    for (uint32_t l = 0; l < k; ++l)
      sum += A[row * k + l] * B[l * n + col];
    C[row * n + col] = sum;
  }
}

void init_matrix(float *mat, uint32_t rows, uint32_t cols) {
  for (uint32_t i = 0; i < rows * cols; ++i)
    mat[i] = (float)rand() / RAND_MAX;
}

double get_time() {
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main(void) {
  float *h_A, *h_B, *h_C_cpu, *h_C_gpu;
  float *d_A, *d_B, *d_C;
  const uint32_t size_A = M * K * sizeof(float);
  const uint32_t size_B = K * N * sizeof(float);
  const uint32_t size_C = M * N * sizeof(float);

  h_A = (float *)malloc(size_A);
  h_B = (float *)malloc(size_B);
  h_C_cpu = (float *)malloc(size_C);
  h_C_gpu = (float *)malloc(size_C);

  srand(time(NULL));
  init_matrix(h_A, M, K);
  init_matrix(h_B, K, N);

  hipMalloc(&d_A, size_A);
  hipMalloc(&d_B, size_B);
  hipMalloc(&d_C, size_C);

  hipMemcpy(d_A, h_A, size_A, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size_B, hipMemcpyHostToDevice);

  const dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
  const dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

  // ------

  const uint32_t warm_up_runs = 100;
  const uint32_t benchmark_runs = 500;

  printf("Performing warm-up runs...\n");
  for (uint32_t i = 0; i < warm_up_runs; ++i) {
    matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);
    matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
    hipDeviceSynchronize();
  }

  printf("Benchmarking CPU implementation...\n");
  double cpu_total_time = 0.0;
  for (uint32_t i = 0; i < benchmark_runs; ++i) {
    double start_time = get_time();

    matmul_cpu(h_A, h_B, h_C_cpu, M, K, N);

    double end_time = get_time();
    cpu_total_time += end_time - start_time;
  }
  double cpu_avg_time = cpu_total_time / benchmark_runs;

  printf("Benchmarking GPU implementation...\n");
  double gpu_total_time = 0.0;
  for (uint32_t i = 0; i < benchmark_runs; ++i) {
    double start_time = get_time();

    matmul_gpu<<<gridDim, blockDim>>>(d_A, d_B, d_C, M, K, N);
    hipDeviceSynchronize();

    double end_time = get_time();
    gpu_total_time += end_time - start_time;
  }
  double gpu_avg_time = gpu_total_time / benchmark_runs;

  printf("CPU average time: %f microseconds\n", (cpu_avg_time * 1e6f));
  printf("GPU average time: %f microseconds\n", (gpu_avg_time * 1e6f));
  printf("Speedup: %fx\n", cpu_avg_time / gpu_avg_time);

  // ------

  free(h_A);
  free(h_B);
  free(h_C_cpu);
  free(h_C_gpu);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  return 0;
}
